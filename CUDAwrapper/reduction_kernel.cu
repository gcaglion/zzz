#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include "../common.h"
#include "reduction_kernel.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */
template <class T> __global__ void reduce0(int what, T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	switch (what) {
	case SUM:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	case SQUARED_SUM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case NORM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case AVERAGE:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	default:
		break;
	}

    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }

        cg::sync(cta);
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/* This version uses contiguous threads, but its interleaved
   addressing results in many shared memory bank conflicts.
*/
template <class T> __global__ void reduce1(int what, T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	switch (what) {
	case SUM:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	case SQUARED_SUM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case NORM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case AVERAGE:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	default:
		break;
	}

    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        int index = 2 * s * tid;

        if (index < blockDim.x)
        {
			switch (what) {
			case SUM:
				sdata[index] += sdata[index+s];
				break;
			case SQUARED_SUM:
				sdata[index] += sdata[index+s]*sdata[index+s];
				break;
			case NORM:
				sdata[index] += sdata[index+s]*sdata[index+s];
				break;
			case AVERAGE:
				sdata[index] += sdata[index+s];
				break;
			default:
				break;
			}
        }

        cg::sync(cta);
    }

    // write result for this block to global mem
	switch (what) {
	case SUM:
		if (tid==0) g_odata[blockIdx.x] = sdata[0];
		break;
	case SQUARED_SUM:
		if (tid==0) g_odata[blockIdx.x] = sdata[0];
		break;
	case NORM:
		if (tid==0) g_odata[blockIdx.x] = sqrtf(sdata[0]);
		break;
	case AVERAGE:
		if (tid==0) g_odata[blockIdx.x] = sdata[0]/n;
		break;
	default:
		break;
	}
}

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
template <class T> __global__ void reduce2(int what, T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	switch (what) {
	case SUM:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	case SQUARED_SUM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case NORM:
		sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case AVERAGE:
		sdata[tid] = (i < n) ? g_idata[i] : 0;
		break;
	default:
		break;
	}

    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
			switch (what) {
			case SUM:
				sdata[tid] += sdata[tid+s];
				break;
			case SQUARED_SUM:
				sdata[tid] += sdata[tid+s]*sdata[tid+s];
				break;
			case NORM:
				sdata[tid] += sdata[tid+s]*sdata[tid+s];
				break;
			case AVERAGE:
				sdata[tid] += sdata[tid+s];
				break;
			default:
				break;
			}
        }

        cg::sync(cta);
    }

    // write result for this block to global mem
	switch (what) {
	case SUM:
		if (tid==0) g_odata[blockIdx.x] = sdata[0];
		break;
	case SQUARED_SUM:
		if (tid==0) g_odata[blockIdx.x] = sdata[0];
		break;
	case NORM:
		if (tid==0) g_odata[blockIdx.x] = sqrtf(sdata[0]);
		break;
	case AVERAGE:
		if (tid==0) g_odata[blockIdx.x] = sdata[0]/n;
		break;
	default:
		break;
	}
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
*/
template <class T> __global__ void reduce3(int what, T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	T mySum;
	switch (what) {
	case SUM:
		mySum = (i<n) ? g_idata[i] : 0;
		break;
	case SQUARED_SUM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case NORM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case AVERAGE:
		mySum = (i<n) ? g_idata[i] : 0;
		break;
	default:
		break;
	}

	if (i+blockDim.x<n) {
		switch (what) {
		case SUM:
			mySum += g_idata[i+blockDim.x];
			break;
		case SQUARED_SUM:
			mySum += g_idata[i+blockDim.x]*g_idata[i+blockDim.x];
			break;
		case NORM:
			mySum += g_idata[i+blockDim.x]*g_idata[i+blockDim.x];
			break;
		case AVERAGE:
			mySum += g_idata[i+blockDim.x];
			break;
		default:
			break;
		}
	}

    sdata[tid] = mySum;
    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
			switch (what) {
			case SUM:
				sdata[tid] = mySum = mySum+sdata[tid+s];
				break;
			case SQUARED_SUM:
				sdata[tid] = mySum = mySum+sdata[tid+s]*sdata[tid+s];
				break;
			case NORM:
				sdata[tid] = mySum = mySum+sdata[tid+s]*sdata[tid+s];
				break;
			case AVERAGE:
				sdata[tid] = mySum = mySum+sdata[tid+s];
				break;
			default:
				break;
			}
        }

        cg::sync(cta);
    }

    // write result for this block to global mem
	switch (what) {
	case SUM:
		if (tid==0) g_odata[blockIdx.x] = mySum;
		break;
	case SQUARED_SUM:
		if (tid==0) g_odata[blockIdx.x] = mySum;
		break;
	case NORM:
		if (tid==0) g_odata[blockIdx.x] = sqrtf(mySum);
		break;
	case AVERAGE:
		if (tid==0) g_odata[blockIdx.x] = mySum/n;
		break;
	default:
		break;
	}
}

/*
    This version uses the warp shuffle operation if available to reduce 
    warp synchronization. When shuffle is not available the final warp's
    worth of work is unrolled to reduce looping overhead.

    See http://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
    for additional information about using shuffle to perform a reduction
    within a warp.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize> __global__ void reduce4(int what, T *g_idata, T *g_odata, unsigned int n) {
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	T mySum;
	switch (what) {
	case SUM:
		mySum = (i<n) ? g_idata[i] : 0;
		break;
	case SQUARED_SUM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case NORM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0;
		break;
	case AVERAGE:
		mySum = (i<n) ? g_idata[i] : 0;
		break;
	default:
		break;
	}
	

	if (i+blockSize<n) {
		switch (what) {
		case SUM:
			mySum += g_idata[i+blockSize];
			break;
		case SQUARED_SUM:
			mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
			break;
		case NORM:
			mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
			break;
		case AVERAGE:
			mySum += g_idata[i+blockSize];
			break;
		default:
			break;
		}
	}

    sdata[tid] = mySum;
    cg::sync(cta);

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
        if (tid < s)
        {
			switch (what) {
			case SUM:
				sdata[tid] = mySum = mySum+sdata[tid+s];
				break;
			case SQUARED_SUM:
				sdata[tid] = mySum = mySum+sdata[tid+s]*sdata[tid+s];
				break;
			case NORM:
				sdata[tid] = mySum = mySum+sdata[tid+s]*sdata[tid+s];
				break;
			case AVERAGE:
				sdata[tid] = mySum = mySum+sdata[tid+s];
				break;
			default:
				break;
			}
        }

        cg::sync(cta);
    }

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
		if (blockSize>=64) {
			switch (what) {
			case SUM:
				mySum += sdata[tid+32];
				break;
			case SQUARED_SUM:
				mySum += sdata[tid+32]*sdata[tid+32];
				break;
			case NORM:
				mySum += sdata[tid+32]*sdata[tid+32];
				break;
			case AVERAGE:
				mySum += sdata[tid+32];
				break;
			default:
				break;
			}
		}
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2) 
        {
			switch (what) {
			case SUM:
				mySum += tile32.shfl_down(mySum, offset); 
				break;
			case SQUARED_SUM:
				mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
				break;
			case NORM:
				mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
				break;
			case AVERAGE:
				mySum += tile32.shfl_down(mySum, offset);
				break;
			default:
				break;
			}			
        }
    }

    // write result for this block to global mem
	switch (what) {
	case SUM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
		break;
	case SQUARED_SUM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
		break;
	case NORM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = sqrtf(mySum);
		break;
	case AVERAGE:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum/n;
		break;
	default:
		break;
	}
}

/*
    This version is completely unrolled, unless warp shuffle is available, then
    shuffle is used within a loop.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time. When shuffle is available, it is used to reduce warp synchronization.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize> __global__ void reduce5(int what, T *g_idata, T *g_odata, unsigned int n)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

	T mySum;
	switch (what) {
	case SUM:
		mySum = (i<n) ? g_idata[i] : 0; 
		break;
	case SQUARED_SUM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0; break;
		break;
	case NORM:
		mySum = (i<n) ? g_idata[i]*g_idata[i] : 0; break;
	case AVERAGE:
		mySum = (i<n) ? g_idata[i] : 0; 
		break;
	default:
		break;
	}

	

	if (i+blockSize<n) {
		switch (what) {
		case SUM:
			mySum += g_idata[i+blockSize];
			break;
		case SQUARED_SUM:
			mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
			break;
		case NORM:
			mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
			break;
		case AVERAGE:
			mySum += g_idata[i+blockSize];
			break;
		default:
			break;
		}
	}
    sdata[tid] = mySum;
    cg::sync(cta);

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+256];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+256]*sdata[tid+256];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+256]*sdata[tid+256];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+256];
			break;
		default:
			break;
		}		
    }

    cg::sync(cta);

    if ((blockSize >= 256) &&(tid < 128))
    {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+128];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+128]*sdata[tid+128];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+128]*sdata[tid+128];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+128];
			break;
		default:
			break;
		}
	}

    cg::sync(cta);

    if ((blockSize >= 128) && (tid <  64))
    {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+64];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+64]*sdata[tid+64];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+64]*sdata[tid+64];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+64];
			break;
		default:
			break;
		}
	}

    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

    if (cta.thread_rank() < 32)
    {
        // Fetch final intermediate sum from 2nd warp
		if (blockSize>=64) {
			switch (what) {
			case SUM:
				sdata[tid] = mySum = mySum+sdata[tid+32];
				break;
			case SQUARED_SUM:
				sdata[tid] = mySum = mySum+sdata[tid+32]*sdata[tid+32];
				break;
			case NORM:
				sdata[tid] = mySum = mySum+sdata[tid+32]*sdata[tid+32];
				break;
			case AVERAGE:
				sdata[tid] = mySum = mySum+sdata[tid+32];
				break;
			default:
				break;
			}
		}
        // Reduce final warp using shuffle
        for (int offset = tile32.size()/2; offset > 0; offset /= 2) 
        {
			switch (what) {
			case SUM:
				mySum += tile32.shfl_down(mySum, offset); 
				break;
			case SQUARED_SUM:
				mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
				break;
			case NORM:
				mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
				break;
			case AVERAGE:
				mySum += tile32.shfl_down(mySum, offset); 
				break;
			default:
				break;
			}			
        }
    }

    // write result for this block to global mem
	switch (what) {
	case SUM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
		break;
	case SQUARED_SUM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
		break;
	case NORM:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = sqrtf(mySum);
		break;
	case AVERAGE:
		if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum/n;
		break;
	default:
		break;
	}
}

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2> __global__ void reduce6(int what, T *g_idata, T *g_odata, unsigned int n) {
	// Handle to thread block group
	cg::thread_block cta = cg::this_thread_block();
	T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2+threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	T mySum = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i<n)
	{
		switch (what) {
		case SUM:
			mySum += g_idata[i];
			break;
		case SQUARED_SUM:
			mySum += g_idata[i]*g_idata[i];
			break;
		case NORM:
			mySum += g_idata[i]*g_idata[i];
			break;
		case AVERAGE:
			mySum += g_idata[i];
			break;
		default:
			break;
		}

		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2||i+blockSize<n)
			switch (what) {
			case SUM:
				mySum += g_idata[i+blockSize];
				break;
			case SQUARED_SUM:
				mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
				break;
			case NORM:
				mySum += g_idata[i+blockSize]*g_idata[i+blockSize];
				break;
			case AVERAGE:
				mySum += g_idata[i+blockSize];
				break;
			default:
				break;
			}

		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = mySum;
	cg::sync(cta);


	// do reduction in shared mem
	if ((blockSize>=512)&&(tid<256)) {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+256];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+256]*sdata[tid+256];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+256]*sdata[tid+256];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+256];
			break;
		default:
			break;
		}

	}

	cg::sync(cta);

	if ((blockSize>=256)&&(tid<128)) {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+128];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+128]*sdata[tid+128];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+128]*sdata[tid+128];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+128];
			break;
		default:
			break;
		}
	}

	cg::sync(cta);

	if ((blockSize>=128)&&(tid<64)) {
		switch (what) {
		case SUM:
			sdata[tid] = mySum = mySum+sdata[tid+64];
			break;
		case SQUARED_SUM:
			sdata[tid] = mySum = mySum+sdata[tid+64]*sdata[tid+64];
			break;
		case NORM:
			sdata[tid] = mySum = mySum+sdata[tid+64]*sdata[tid+64];
			break;
		case AVERAGE:
			sdata[tid] = mySum = mySum+sdata[tid+64];
			break;
		default:
			break;
		}
	}

	cg::sync(cta);

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	if (cta.thread_rank()<32) {
		// Fetch final intermediate sum from 2nd warp
		if (blockSize>=64) {
			switch (what) {
			case SUM:
				sdata[tid] = mySum = mySum+sdata[tid+32];
				break;
			case SQUARED_SUM:
				sdata[tid] = mySum = mySum+sdata[tid+32]*sdata[tid+32];
				break;
			case NORM:
				sdata[tid] = mySum = mySum+sdata[tid+32]*sdata[tid+32];
				break;
			case AVERAGE:
				sdata[tid] = mySum = mySum+sdata[tid+32];
				break;
			default:
				break;
			}
			// Reduce final warp using shuffle
			for (int offset = tile32.size()/2; offset>0; offset /= 2)
			{
				switch (what) {
				case SUM:
					mySum += tile32.shfl_down(mySum, offset);
					break;
				case SQUARED_SUM:
					mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
					break;
				case NORM:
					mySum += tile32.shfl_down(mySum, offset)*tile32.shfl_down(mySum, offset);
					break;
				case AVERAGE:
					mySum += tile32.shfl_down(mySum, offset);
					break;
				default:
					break;
				}
			}
		}

		// write result for this block to global mem
		switch (what) {
		case SUM:
			if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
			break;
		case SQUARED_SUM:
			if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum;
			break;
		case NORM:
			if (cta.thread_rank()==0) g_odata[blockIdx.x] = sqrtf(mySum);
			break;
		case AVERAGE:
			if (cta.thread_rank()==0) g_odata[blockIdx.x] = mySum/n;
			break;
		default:
			break;
		}

	}
}

bool isPow2(unsigned int x) {
	return ((x&(x-1))==0);
}


////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T> void reduce(int what, int size, int threads, int blocks, int whichKernel, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    // choose which of the optimized versions of reduction to launch
    switch (whichKernel)
    {
        case 0:
            reduce0<T><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
            break;

        case 1:
            reduce1<T><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
            break;

        case 2:
            reduce2<T><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
            break;

        case 3:
            reduce3<T><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
            break;

        case 4:
            switch (threads)
            {
                case 512:
                    reduce4<T, 512><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 256:
                    reduce4<T, 256><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 128:
                    reduce4<T, 128><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 64:
                    reduce4<T,  64><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 32:
                    reduce4<T,  32><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 16:
                    reduce4<T,  16><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  8:
                    reduce4<T,   8><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  4:
                    reduce4<T,   4><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  2:
                    reduce4<T,   2><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  1:
                    reduce4<T,   1><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;
            }

            break;

        case 5:
            switch (threads)
            {
                case 512:
                    reduce5<T, 512><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 256:
                    reduce5<T, 256><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 128:
                    reduce5<T, 128><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 64:
                    reduce5<T,  64><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 32:
                    reduce5<T,  32><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case 16:
                    reduce5<T,  16><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  8:
                    reduce5<T,   8><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  4:
                    reduce5<T,   4><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  2:
                    reduce5<T,   2><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;

                case  1:
                    reduce5<T,   1><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                    break;
            }

            break;

        case 6:
        default:
            if (isPow2(size))
            {
                switch (threads)
                {
                    case 512:
                        reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 256:
                        reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 128:
                        reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 64:
                        reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 32:
                        reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 16:
                        reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  8:
                        reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  4:
                        reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  2:
                        reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  1:
                        reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;
                }
            }
            else
            {
                switch (threads)
                {
                    case 512:
                        reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 256:
                        reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 128:
                        reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 64:
                        reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 32:
                        reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case 16:
                        reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  8:
                        reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  4:
                        reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  2:
                        reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;

                    case  1:
                        reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(what, d_idata, d_odata, size);
                        break;
                }
            }

            break;
    }
}
// Instantiate the reduction function for 3 types
template void reduce<int>(int what, int size, int threads, int blocks, int whichKernel, int *d_idata, int *d_odata);
template void reduce<float>(int what, int size, int threads, int blocks, int whichKernel, float *d_idata, float *d_odata);
EXPORT void reduce(int what, int size, int threads, int blocks, int whichKernel, float *d_idata, float *d_odata) {
	reduce<float>(what, size, threads, blocks, whichKernel, d_idata, d_odata);
}
template void reduce<double>(int what, int size, int threads, int blocks, int whichKernel, double *d_idata, double *d_odata);

#endif // #ifndef _REDUCE_KERNEL_H_
