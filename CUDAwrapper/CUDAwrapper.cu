#include "hip/hip_runtime.h"
#include "CUDAwrapper.h"
//--
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <stdio.h>
//--

void swap(int* v1, int* v2) {
	int tmp=(*v1);
	(*v1)=(*v2);
	(*v2)=tmp;
}

static const char *cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}

EXPORT void initCUDA() {
	// init CUDA GPU
	int ret=hipSetDevice(0);
	if (ret!=hipSuccess) CUWfail("hipSetDevice failed!  Error %d . Do you have a CUDA-capable GPU installed?\n", ret);
}
EXPORT void initCUBLAS(void* cublasH) {

	if (hipblasCreate((hipblasHandle_t*)cublasH)!=HIPBLAS_STATUS_SUCCESS) CUWfail("CUBLAS initialization error!\n");

}
EXPORT void initCURand(void* cuRandH) {
	if (hiprandCreateGenerator((hiprandGenerator_t*)cuRandH, HIPRAND_RNG_PSEUDO_DEFAULT)!=HIPRAND_STATUS_SUCCESS) {
		//if (hiprandCreateGenerator((hiprandGenerator_t*)cuRandH, HIPRAND_RNG_PSEUDO_DEFAULT)!=HIPRAND_STATUS_SUCCESS) {
		CUWfail("CURAND initialization error!\n");
	}
	/* Set seed */
	if (hiprandSetPseudoRandomGeneratorSeed((*(hiprandGenerator_t*)cuRandH), timeGetTime())!=HIPRAND_STATUS_SUCCESS) {
		CUWfail("CURAND initialization error!\n");
	}
}
EXPORT void initCUstreams(void* cuStream[]) {
	for (int s=0; s<MAX_STREAMS; s++) {
		if (hipStreamCreate((hipStream_t*)cuStream[s])!=hipSuccess) CUWfail("CU stream %d creation failed.", s);
	}
}

EXPORT void Malloc_cu(numtype** var, int size) {
	if (hipMalloc(var, size*sizeof(numtype))!=hipSuccess) CUWfail("F41LUR3!-1111");
}
EXPORT void Free_cu(numtype* var) {
	if (hipFree(var)!=hipSuccess) CUWfail("F41LUR3!-2222");
}

//-- CPU<->GPU transfer functions
/*EXPORT void h2d_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if(MAX_STREAMS==0) {
		if (hipMemcpy(destAddr, srcAddr, size, hipMemcpyHostToDevice)!=hipSuccess) CUWfail("F41LUR3!-333")
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyHostToDevice, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				CUWfail("s=%d ; CUDA error %d\n", s, hipGetLastError());
			}
		}
	}
}
EXPORT void d2h_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if(MAX_STREAMS==0) {
		if(hipMemcpy(destAddr, srcAddr, size, hipMemcpyDeviceToHost)!=hipSuccess) CUWfail("F41LUR3!-444")
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyDeviceToHost, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				CUWfail("s=%d ; CUDA error %d\n", s, hipGetLastError());
			}
		}
	}
}
*/
EXPORT void h2d_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if (cuStream==nullptr) {
		if (!(hipMemcpy(destAddr, srcAddr, size, hipMemcpyHostToDevice)==hipSuccess)) CUWfail("CUDA error %d", hipGetLastError());
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyHostToDevice, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				CUWfail("s=%d ; CUDA error %d\n", s, hipGetLastError());
			}
		}
	}
}
EXPORT void d2h_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	if (cuStream==nullptr) {
		if (!(hipMemcpy(destAddr, srcAddr, size, hipMemcpyDeviceToHost)==hipSuccess))  CUWfail("CUDA error %d", hipGetLastError());
	} else {
		int streamSize=size/sizeof(numtype)/MAX_STREAMS;
		size_t streamBytes=streamSize*sizeof(numtype);
		for (int s=0; s<MAX_STREAMS; s++) {
			int offset=s*streamSize;
			if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyDeviceToHost, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
				CUWfail("s=%d ; CUDA error %d\n", s, hipGetLastError());
			}
		}
	}
}

//==================================
__global__	void initGPUData_ker(float *data, int numElements, float value) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < numElements) {
		data[tid] = value;
	}
}
EXPORT		void initGPUData(float *data, int numElements, float value) {
	dim3 gridDim;
	dim3 blockDim;

	blockDim.x = 1024;
	gridDim.x = (numElements+blockDim.x-1)/blockDim.x;

	initGPUData_ker<<< gridDim, blockDim>>> (data, numElements, value);
}

EXPORT bool loadBatchData_cu(numtype* destAddr, numtype* srcAddr, int size, void* cuStream[]) {
	int streamSize=size/sizeof(numtype)/MAX_STREAMS;
	size_t streamBytes=streamSize*sizeof(numtype);
	for (int s=0; s<MAX_STREAMS; s++) {
		int offset=s*streamSize;
		if (hipMemcpyAsync(&destAddr[offset], &srcAddr[offset], streamBytes, hipMemcpyHostToDevice, (*(hipStream_t*)cuStream[s]))!=hipSuccess) {
			printf("s=%d ; CUDA error %d\n", s, hipGetLastError());
			return false;
		}
	}
	return true;
}
EXPORT bool dumpArray_cu(int vlen, numtype* v, const char* fname) {
	numtype* hw=(numtype*)malloc(vlen*sizeof(numtype));
	if (hipMemcpy(hw, v, vlen*sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return false;
	FILE* f=fopen(fname, "w");
	if (f==nullptr) return false;
	for (int i=0; i<vlen; i++) fprintf(f, "%f\n", hw[i]);
	free(hw);
	fclose(f);
	return true;
}
EXPORT bool loadArray_cu(int vlen, numtype* v, const char* fname){
	numtype fh;
	numtype* vh=(numtype*)malloc(vlen*sizeof(numtype));
	FILE* f=fopen(fname, "r");
	if (f==nullptr) return false;
	for (int i=0; i<vlen; i++) {
		if(fscanf(f, "%f\n", &fh)==0) return false;
		vh[i]=fh;
	}
	if (hipMemcpy(v, vh, vlen*sizeof(numtype), hipMemcpyHostToDevice)!=hipSuccess) return false;
	fclose(f);
	free(vh);
	return true;
}

//-- matrix functions
EXPORT bool cuMtr_cublas(void* cublasH, int my, int mx, numtype* m, numtype* otm) {
	float alpha=1;
	float beta=0;
	if (hipblasSgeam((*(hipblasHandle_t*)cublasH), HIPBLAS_OP_T, HIPBLAS_OP_T, my, mx, &alpha, m, mx, &beta, m, mx, otm, my)!=HIPBLAS_STATUS_SUCCESS) return false;
	return true;
}

EXPORT bool MbyM_cu(void* cublasH, int Ay, int Ax, numtype Ascale, bool Atr, numtype* A, int By, int Bx, numtype Bscale, bool Btr, numtype* B, numtype* C) {

	float *alpha = &Ascale;
	float *beta = &Bscale;

	hipblasOperation_t Aop=HIPBLAS_OP_N;
	hipblasOperation_t Bop=HIPBLAS_OP_N;
	int m=Bx;
	int n=Ay;
	int k=Ax;
	int ldA=Ax;
	int ldB=Bx;
	int ldC=Bx;

	numtype* vA = A;
	numtype* vB = B;

	if (Atr) {
		Aop=HIPBLAS_OP_T;
		n=Ax; k=Ay;
	}
	if (Btr) {
		Bop=HIPBLAS_OP_T;
		m=By;
		ldC=By;
	}

	if (!Vinit_cu(m*n, C, 0, 0)) return false;
	if (hipblasSgemm((*(hipblasHandle_t*)cublasH), Bop, Aop, m, n, k, alpha, vB, ldB, vA, ldA, beta, C, ldC)!=HIPBLAS_STATUS_SUCCESS) throw(new std::exception("call to cublasSgem()"));

	return true;
}

__global__ void VdotV_ker(int n, float x[], float y[], float* dot_p) {
	float tmp;
	int i = blockDim.x * blockIdx.x+threadIdx.x;

	if (i < n) {
		tmp = x[i]*y[i];
		atomicAdd(dot_p, tmp);
	}
}
__global__ void cuSadd(const numtype* s1, const numtype* s2, numtype* ssum) {
	ssum[0]=s1[0]+s2[0];
}
__global__ void cuVscale_ker(const int vlen, numtype *v, const numtype s) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) v[tid] *= s;
}
__global__ void cuVcopy_ker(const int vlen, const numtype *v1, numtype *v2) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) v2[tid] = v1[tid];
}
__global__ void cuVminusV_ker(const int vlen, const numtype *a, const numtype sa, const numtype *b, const numtype sb, numtype* c) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) c[tid] = a[tid]*sa-b[tid]*sb;
}
__global__ void cuVplusV_ker(const int vlen, const numtype *a, const numtype sa, const numtype *b, const numtype sb, numtype* c) {
	int tid = blockIdx.x * blockDim.x+threadIdx.x;
	if (tid < vlen) c[tid] = a[tid]*sa+b[tid]*sb;
}
__global__ void cuVsum_ker(const int vlen, const numtype *v, numtype* osum) {

	//@@ Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*CUDA_BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2*blockIdx.x * CUDA_BLOCK_SIZE;
	if (start+t < vlen)
		partialSum[t] = v[start+t];
	else
		partialSum[t] = 0;
	if (start+CUDA_BLOCK_SIZE+t < vlen)
		partialSum[CUDA_BLOCK_SIZE+t] = v[start+CUDA_BLOCK_SIZE+t];
	else
		partialSum[CUDA_BLOCK_SIZE+t] = 0;
	//@@ Traverse the reduction tree
	for (unsigned int stride = CUDA_BLOCK_SIZE; stride>=1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t+stride];
	}
	//@@ Write the computed sum of the block to the output vector at the 
	//@@ correct index
	if (t==0)
		osum[blockIdx.x] = partialSum[0];

}
__global__ void cuVssum_ker(const int vlen, const numtype *v, numtype* ossum) {

	//@@ Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*CUDA_BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2*blockIdx.x * CUDA_BLOCK_SIZE;
	if (start+t < vlen)
		partialSum[t] = v[start+t]*v[start+t];
	else
		partialSum[t] = 0;
	if (start+CUDA_BLOCK_SIZE+t < vlen)
		partialSum[CUDA_BLOCK_SIZE+t] = v[start+CUDA_BLOCK_SIZE+t]*v[start+CUDA_BLOCK_SIZE+t];
	else
		partialSum[CUDA_BLOCK_SIZE+t] = 0;
	//@@ Traverse the reduction tree
	for (unsigned int stride = CUDA_BLOCK_SIZE; stride>=1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t+stride];
	}
	//@@ Write the computed sum of the block to the output vector at the 
	//@@ correct index
	if (t==0)
		ossum[blockIdx.x] = partialSum[0];

}
__global__ void Vscale(int vlen, numtype* v, numtype scaleM, numtype scaleP) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) v[i] = scaleM*v[i]+scaleP;
}
__global__ void Vinit_ker(int vlen, numtype* v, numtype start, numtype inc) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) v[i] = start+i*inc;
}
__global__ void VbyV2V_ker(int vlen, numtype* v1, numtype* v2, numtype* ov) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) ov[i]=v1[i]*v2[i];
}

//-- scalar functions
EXPORT bool Sadd_cu(numtype* s1, numtype* s2, numtype* ssum) {
	cuSadd<<< 1, 1>>>(s1, s2, ssum);
	return ((hipGetLastError()==hipSuccess));
}

//-- vector functions;
EXPORT bool getMcol_cu(void* cublasH, int Ay, int Ax, numtype* A, int col, numtype* oCol) {
	hipblasStatus_t err=hipblasScopy((*((hipblasHandle_t*)cublasH)), Ax, A, Ax, oCol, 1);
	if (err!=HIPBLAS_STATUS_SUCCESS) {
		printf("getMcol_cu() CUBLAS error %d: %s\n", err, cudaGetErrorEnum(err));
		return false;
	}
	return true;
}
EXPORT bool Vscale_cu(int vlen, numtype* v, numtype s){
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVscale_ker<<< gridDim, blockDim>>> (vlen, v, s);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Vcopy_cu(int vlen, numtype* v1, numtype* v2) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVcopy_ker<<< gridDim, blockDim>>> (vlen, v1, v2);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Vadd_cu(int vlen, numtype* v1, numtype scale1, numtype* v2, numtype scale2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVplusV_ker<<< gridDim, blockDim>>> (vlen, v1, scale1, v2, scale2, ov);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Vdiff_cu(int vlen, numtype* v1, numtype scale1, numtype* v2, numtype scale2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVminusV_ker<<< gridDim, blockDim>>> (vlen, v1, scale1, v2, scale2, ov);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Vsum_cu(int vlen, numtype* v, numtype* ovsum, numtype* ss_d) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVsum_ker<<< gridDim, blockDim>>> (vlen, v, ss_d );

	if (hipMemcpy(ovsum, ss_d, sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return false;

	return ((hipGetLastError()==hipSuccess));
}

EXPORT void VdotV_cu(int n, float x_d[], float y_d[], float* dot_d, int blocks, int threads) {

	hipMemset(dot_d, 0, sizeof(float));

	/* Invoke kernel */
	VdotV_ker<<<blocks, threads>>>(n, x_d, y_d, dot_d);

	//hipMemcpy(oVdotVh, dot_d, sizeof(float), hipMemcpyDeviceToHost);

}

/*EXPORT bool VdotV_cu(int vlen, numtype* v1, numtype* v2, numtype* ovdotv) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	VdotV_ker<<< gridDim, blockDim>>> (vlen, v1, v2, ovdotv);

	return ((hipGetLastError()==hipSuccess));
}
*/
EXPORT bool Vssum_cu(int vlen, numtype* v, numtype* ovssum) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuVssum_ker<<< gridDim, blockDim>>> (vlen, v, ovssum);

	return ((hipGetLastError()==hipSuccess));
}
EXPORT bool Vssum_cu_cublas(void* cublasH, int Vlen, numtype* V, numtype* oVssum, numtype* ss_d) {
	if (hipblasSnrm2((*(hipblasHandle_t*)cublasH), Vlen, V, 1, oVssum)!=HIPBLAS_STATUS_SUCCESS) return false;
	(*oVssum)=(*oVssum)*(*oVssum);
	return true;
}

EXPORT bool Vnorm_cu(void* cublasH, int Vlen, numtype* V,  numtype* oVnorm, numtype* ss_d) {
	if (hipblasSnrm2((*(hipblasHandle_t*)cublasH), Vlen, V, 1, oVnorm)!=HIPBLAS_STATUS_SUCCESS) return false;
	//if (hipMemcpy(oVnorm, ss_d, sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return false;
	return true;
}
EXPORT bool Vinit_cu(int vlen, numtype* v, numtype start, numtype inc) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	Vinit_ker<<< gridDim, blockDim>>> (vlen, v, start, inc);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool VbyV2V_cu(int vlen, numtype* v1, numtype* v2, numtype* ov) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	VbyV2V_ker<<< gridDim, blockDim>>> (vlen, v1, v2, ov);

	return((hipGetLastError()==hipSuccess));
}

EXPORT bool VinitRnd_cu(int vlen, numtype* v, numtype rndmin, numtype rndmax, void* cuRandH) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	//-- Generate n floats on device, with  values between 0.0 and 1.0, where 0.0 is excluded and 1.0 is included
	if(hiprandGenerateUniform((*(hiprandGenerator_t*)cuRandH), v, vlen) !=HIPRAND_STATUS_SUCCESS) return false;
	//-- need to scale to rndmin<->rndmax
	Vscale<<< gridDim, blockDim>>>(vlen, v, (rndmax-rndmin), rndmax-(rndmax-rndmin)*1);

	/*/-- !!!!!!!!!!!!! REMOVE !!!!!!!!!!
	numtype* hw=(numtype*)malloc(vlen*sizeof(numtype));
	if (hipMemcpy(hw, v, vlen*sizeof(numtype), hipMemcpyDeviceToHost)!=hipSuccess) return false;
	char* fname = "C:/temp/rndw.txt";
	FILE* f=fopen(fname, "w");
	for (int i=0; i<vlen; i++) fprintf(f, "%f\n", hw[i]);
	free(hw);
	fclose(f);
	//--
	*/
	return((hipGetLastError()==hipSuccess));
}

__global__ void cuTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = tanhf(in[i]);
}
__global__ void cudTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1-tanhf(in[i])*tanhf(in[i]);
}
__global__ void ORIG_cuTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) out[i] = tanhf(in[i]);
}
__global__ void ORIG_cudTanh_ker(int vlen, numtype* in, numtype* out) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<vlen) out[i] = 1-tanhf(in[i])*tanhf(in[i]);
}
__global__ void cuExp4_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1/(1+exp(-4*in[i]));
}
__global__ void cudExp4_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 4*exp(4*in[i])/(pow(exp(4*in[i])+1, 2));
}
__global__ void cuRelu_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = ((in[i] > 0) ? 1 : 0);
}
__global__ void cudRelu_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = ((in[i] > 0) ? in[i] : 0);
}
__global__ void cuSoftPlus_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = log(1+exp(in[i]));
}
__global__ void cudSoftPlus_ker(int vlen, numtype* in, numtype* out) {
	int i = threadIdx.x+blockIdx.x * blockDim.x;
	out[i] = 1/(1+exp(-in[i]));
}

EXPORT bool Tanh_cu(int vlen, numtype* in, numtype* out) {
	/*	int blockSize=64; // The launch configurator returned block size
	int minGridSize; // The minimum grid size needed to achieve the // maximum occupancy for a full device
	int gridSize; // The actual grid size needed, based on input // size
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cudTanh_ker, 0, vlen);
	// Round up according to array size
	gridSize = (vlen+blockSize-1)/blockSize;
	cudTanh_ker<<< gridSize, blockSize>>> (vlen, in, out);
	*/
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuTanh_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool dTanh_cu(int vlen, numtype* in, numtype* out) {
/*	int blockSize=64; // The launch configurator returned block size
	int minGridSize; // The minimum grid size needed to achieve the // maximum occupancy for a full device 
	int gridSize; // The actual grid size needed, based on input // size 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cudTanh_ker, 0, vlen);
	// Round up according to array size 
	gridSize = (vlen+blockSize-1)/blockSize;
	cudTanh_ker<<< gridSize, blockSize>>> (vlen, in, out);
*/
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudTanh_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Exp4_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;
	cuExp4_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool dExp4_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudExp4_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool Relu_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuRelu_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool dRelu_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudRelu_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool SoftPlus_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cuSoftPlus_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
EXPORT bool dSoftPlus_cu(int vlen, numtype* in, numtype* out) {
	dim3 gridDim;
	dim3 blockDim;
	blockDim.x = CUDA_BLOCK_SIZE;
	gridDim.x = (vlen+blockDim.x-1)/blockDim.x;

	cudSoftPlus_ker<<< gridDim, blockDim>>> (vlen, in, out);

	return((hipGetLastError()==hipSuccess));
}
